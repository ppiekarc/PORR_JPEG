#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include "../ycc_converter.h"
#include "dctCUDAv2.h"
#include <hip/hip_runtime.h>
#include <>
#include "hip/hip_runtime_api.h"
#include "helper_timer.h"

#define N 64
#define Y 0
#define Cb 1
#define Cr 2

__constant__ static uint8_t zigzag[64] = {
	0, 1, 5, 6,14,15,27,28,
	2, 4, 7,13,16,26,29,42,
	3, 8,12,17,25,30,41,43,
	9,11,18,24,31,40,44,53,
	10,19,23,32,39,45,52,54,
	20,22,33,38,46,51,55,60,
	21,34,37,47,50,56,59,61,
	35,36,48,49,57,58,62,63
};

__constant__ static float fdtbl_Y[N];
__constant__ static float fdtbl_Cb[N];


#define image_(t, index) image[(t * width * height) + index]
#define result_(t, b, p) result[(t * width * height) + (b * 64) + p]

__global__ static void dtf_kernel(int16_t *result, int8_t *image, size_t width, size_t height)
{

	int type = blockIdx.z;
	//uint8_t block_nr = blockIdx.x + (gridDim.x * blockIdx.y);
	
	float tmp0, tmp1, tmp2, tmp3, tmp4, tmp5, tmp6, tmp7;
	float tmp10, tmp11, tmp12, tmp13;
	float z1, z2, z3, z4, z5, z11, z13;
	float *dataptr;
	__shared__ float datafloat[64];
	float temp;
	float *fdtbl;
	int8_t ctr;
	size_t j = threadIdx.x;
	size_t i = threadIdx.y;
	//printf("jestesmy w watku bx = %d by = %d bz = %d tx = %d ty=%d \n",
	//	blockIdx.x, blockIdx.y, type, threadIdx.x, threadIdx.y);

	/*
	for (size_t i = 0; i < 8; i++) {
		for (size_t j = 0; j < 8; j++) {
			int index = (blockIdx.x * 8) + i + (width * blockIdx.y * 8) + width * j;
			
			datafloat[i + (8 * j)] = image_(type, index);
		}
	}
	*/
	if (i == 0) {
	for (size_t i = 0; i < 8; i++) {
			int index = (blockIdx.x * 8) + i + (width * blockIdx.y * 8) + width * j;
			datafloat[i + (8 * j)] = image_(type, index);
	}



	/* Pass 1: process rows. */
	//dataptr = datafloat;
			
				dataptr = datafloat + (j * 8);
				//for (ctr = 7; ctr >= 0; ctr--) {
				tmp0 = dataptr[0] + dataptr[7];
				tmp7 = dataptr[0] - dataptr[7];
				tmp1 = dataptr[1] + dataptr[6];
				tmp6 = dataptr[1] - dataptr[6];
				tmp2 = dataptr[2] + dataptr[5];
				tmp5 = dataptr[2] - dataptr[5];
				tmp3 = dataptr[3] + dataptr[4];
				tmp4 = dataptr[3] - dataptr[4];

				tmp10 = tmp0 + tmp3;
				tmp13 = tmp0 - tmp3;
				tmp11 = tmp1 + tmp2;
				tmp12 = tmp1 - tmp2;

				dataptr[0] = tmp10 + tmp11;
				dataptr[4] = tmp10 - tmp11;

				z1 = (tmp12 + tmp13) * ((float) 0.707106781);
				dataptr[2] = tmp13 + z1;
				dataptr[6] = tmp13 - z1;

				tmp10 = tmp4 + tmp5;
				tmp11 = tmp5 + tmp6;
				tmp12 = tmp6 + tmp7;

				z5 = (tmp10 - tmp12) * ((float) 0.382683433);
				z2 = ((float) 0.541196100) * tmp10 + z5;
				z4 = ((float) 1.306562965) * tmp12 + z5;
				z3 = tmp11 * ((float) 0.707106781);

				z11 = tmp7 + z3;
				z13 = tmp7 - z3;

				dataptr[5] = z13 + z2;
				dataptr[3] = z13 - z2;
				dataptr[1] = z11 + z4;
				dataptr[7] = z11 - z4;
				//dataptr += 8;
			//}
			}
	/* Pass 2: process columns */
	//dataptr = datafloat;
			if (i == 1) {
				dataptr = datafloat + j;

				//for (ctr = 7; ctr >= 0; ctr--) {
				tmp0 = dataptr[0] + dataptr[56];
				tmp7 = dataptr[0] - dataptr[56];
				tmp1 = dataptr[8] + dataptr[48];
				tmp6 = dataptr[8] - dataptr[48];
				tmp2 = dataptr[16] + dataptr[40];
				tmp5 = dataptr[16] - dataptr[40];
				tmp3 = dataptr[24] + dataptr[32];
				tmp4 = dataptr[24] - dataptr[32];

				tmp10 = tmp0 + tmp3;
				tmp13 = tmp0 - tmp3;
				tmp11 = tmp1 + tmp2;
				tmp12 = tmp1 - tmp2;

				dataptr[0] = tmp10 + tmp11;
				dataptr[32] = tmp10 - tmp11;

				z1 = (tmp12 + tmp13) * ((float) 0.707106781);
				dataptr[16] = tmp13 + z1;
				dataptr[48] = tmp13 - z1;

				tmp10 = tmp4 + tmp5;
				tmp11 = tmp5 + tmp6;
				tmp12 = tmp6 + tmp7;

				z5 = (tmp10 - tmp12) * ((float) 0.382683433);
				z2 = ((float) 0.541196100) * tmp10 + z5;
				z4 = ((float) 1.306562965) * tmp12 + z5;
				z3 = tmp11 * ((float) 0.707106781);

				z11 = tmp7 + z3;
				z13 = tmp7 - z3;
				dataptr[40] = z13 + z2;
				dataptr[24] = z13 - z2;
				dataptr[8] = z11 + z4;
				dataptr[56] = z11 - z4;

				//dataptr++;
			//}

				fdtbl = (type == Y) ? fdtbl_Y : fdtbl_Cb;


				for (size_t i = 0; i < 8; i++) {
					//for (size_t j = 0; j < 8; j++) {
						/* quantization and scaling factor */
					temp = datafloat[i + (8 * j)] * fdtbl[i + (8 * j)];
					/* Round to nearest integer. */
					//result_(type, block_nr, zigzag[i]) = (int16_t)((int16_t)(temp + 16384.5) - 16384);
					//result_(type, block_nr, i) = (int16_t)((int16_t)(d[i] + 16384.5) - 16384);


					size_t block_nr = blockIdx.x + (gridDim.x * blockIdx.y);
					size_t pixel_nr = zigzag[i + 8 * j];
					//int index = (blockIdx.x * 8) + i + (width * blockIdx.y * 8) + width * j;
					result_(type, block_nr, pixel_nr) = (int16_t)((int16_t)(temp + 16384.5) - 16384);
					//result_(type, block_nr, (i + 8 * j)) = image_(type, index);
				}
			}
			
	//}
}


int16_t *dct_CUDAv2(int8_t *Y_in, int8_t *Cb_in, int8_t *Cr_in, size_t width, size_t height, int *num_blocks, const float *dtY, const float *dtCb)
{
	int8_t *dev_image; /* zawiera 3 skladowe obrazka (Y, Cb, Cr)*/

					   /* zmienne zawierja tablice 3 elemntowa dla 3 skladowych obrazka (Y, Cb, Cr)
					   kazdej skladowej przypozadkowana jest tablica blokow
					   a w kazdy blok jest tablica 64 elementowa	*/
	int16_t *dev_res;
	int16_t *host_result;

	hipEvent_t start, stop; // pomiar czasu wykonania j�dra
	float elapsedTime = 0.0f;


	int grid_size_x = (int)(width / 8); /* liczba blokow watkow w sieci w kierunku x */
	int grid_size_y = (int)(height / 8); /* liczba blokow watkow w sieci w kierunku y */

	/* rozmiar siatki grid_size_x * grid_size_y * 3 skladowe obrazka (Y, Cb, Cr) */
	dim3 dimGrid(grid_size_x, grid_size_y, 3);
	/* rozmiar bloku watkow, zawieral bedzie 1 blok do obliczenia DCT*/
	//dim3 dimBlock(1, 1);
	dim3 dimBlock(8, 2);
	/* rozmiar pamieci wspoldzielonej przez 1 blok watkow */
	size_t shShize = (64 * sizeof(int8_t));

	checkCudaErrors(hipSetDevice(0));

	/* Alokacja pamieci */

	/* alokacja danych wejsciowych do urzadzenia */
	checkCudaErrors(hipMalloc((void **)&dev_image, 3 * (width * height * sizeof(int8_t))));

	/* alokacja dla danych wyjsciowych z urzadzenia*/
	checkCudaErrors(hipMalloc((void **)&dev_res, 3 * (width * height * sizeof(int16_t))));

	/* alokacj dla danych wyjsciowych do hosta */
	host_result = (int16_t *)malloc(3 * width * height * sizeof(int16_t));

	/* przekopiowanie stalych tablic kwantyzacji do urzadzenia */
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(fdtbl_Y), (void *)dtY, (N) * sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(fdtbl_Cb), (void *)dtCb, (N) * sizeof(float)));

	/* kopiowanie pami�ci do urz�dzenia */
	checkCudaErrors(hipMemcpy(dev_image + (width * height * Y), Y_in, width * height * sizeof(int8_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_image + (width * height * Cb), Cb_in, width * height * sizeof(int8_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_image + (width * height * Cr), Cr_in, width * height * sizeof(int8_t), hipMemcpyHostToDevice));

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));


	/* wywolanie funkcji jadra */
	dtf_kernel << < dimGrid, dimBlock, shShize >> > (dev_res, dev_image, width, height);

	checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipDeviceSynchronize());


	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

	printf("GPU (kernel) time = %.3f ms \n",
		elapsedTime);

	/* Kopiowanie wynikow z pamieci urzadzenia do hosta */
	checkCudaErrors(hipMemcpy(host_result, dev_res, 3 * width * height * sizeof(int16_t), hipMemcpyDeviceToHost));
	

	/* zwolnienie pamieci */

	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));
	hipFree(dev_image);
	hipFree(dev_res);

	checkCudaErrors(hipDeviceReset());

	return host_result;
}