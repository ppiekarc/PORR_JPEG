#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include "../ycc_converter.h"
#include "dctCUDA.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "helper_timer.h"
#include "../huffman.h"

#define N 64
#define Y 0
#define Cb 1
#define Cr 2

__constant__ static uint8_t zigzag[64] = { 
		0, 1, 5, 6,14,15,27,28,
		2, 4, 7,13,16,26,29,42,
		3, 8,12,17,25,30,41,43,
		9,11,18,24,31,40,44,53,
		10,19,23,32,39,45,52,54,
		20,22,33,38,46,51,55,60,
		21,34,37,47,50,56,59,61,
		35,36,48,49,57,58,62,63 
};

__constant__ static float cos_table[8][8] = {
	{ 1.0000, 0.9808, 0.9239, 0.8315, 0.7071, 0.5556, 0.3827, 0.1951 },
	{ 1.0000, 0.8315, 0.3827, -0.1951, -0.7071, -0.9808, -0.9239, -0.5556 },
	{ 1.0000, 0.5556 ,-0.3827 ,-0.9808 ,-0.7071, 0.1951, 0.9239, 0.8315 },
	{ 1.0000, 0.1951, -0.9239, -0.5556, 0.7071, 0.8315, -0.3827, -0.9808 },
	{ 1.0000, -0.1951, -0.9239, 0.5556, 0.7071, -0.8315, -0.3827, 0.9808 },
	{ 1.0000, -0.5556, -0.3827, 0.9808, -0.7071, -0.1951, 0.9239, -0.8315 },
	{ 1.0000, -0.8315, 0.3827, 0.1951, -0.7071, 0.9808, -0.9239, 0.5556 },
	{ 1.0000, -0.9808, 0.9239, -0.8315, 0.7071, -0.5556, 0.3827, -0.1951 }
};

//__constant__ static uint8_t fdtbl_Y[64] = {
//	16,  11,  10,  16,  24,  40,  51,  61,
//	12,  12,  14,  19,  26,  58,  60,  55,
//	14,  13,  16,  24,  40,  57,  69,  56,
//	14,  17,  22,  29,  51,  87,  80,  62,
//	18,  22,  37,  56,  68, 109, 103,  77,
//	24,  35,  55,  64,  81, 104, 113,  92,
//	49,  64,  78,  87, 103, 121, 120, 101,
//	72,  92,  95,  98, 112, 100, 103,  99
//};
//__constant__ static uint8_t fdtbl_Cb[64] = {
//	17,  18,  24,  47,  99,  99,  99,  99,
//	18,  21,  26,  66,  99,  99,  99,  99,
//	24,  26,  56,  99,  99,  99,  99,  99,
//	47,  66,  99,  99,  99,  99,  99,  99,
//	99,  99,  99,  99,  99,  99,  99,  99,
//	99,  99,  99,  99,  99,  99,  99,  99,
//	99,  99,  99,  99,  99,  99,  99,  99,
//	99,  99,  99,  99,  99,  99,  99,  99
//};

__constant__ static uint8_t fdtbl_Y[64];
__constant__ static uint8_t fdtbl_Cb[64];


#define alpha(u) ((u == 0) ? 1 / sqrt(8.0f) : 0.5f)
#define image_(t, index) image[(t * nr_p) + index]
#define result_(t, b, p) result[(t * nr_p) + (b * 64) + p]

__global__ static void dtf_kernel(int16_t *result, int8_t *image, size_t width, size_t height, size_t nr_p)
{
	__shared__ int8_t block_in[N];
	int16_t converted;
	int i = threadIdx.x;
	int j = threadIdx.y;
	/* w polu type zawarta informacja krorej skladowej dotycza obliczenia : Y Cb Cr */
	int type = blockIdx.z;

	/* wyliczone wartosci indexow tak aby podzielic na odpowiednie bloki, 
		aby kazdy watek wzial odpowiednia dla siebie probke	*/
	int index = (blockIdx.x * blockDim.x) + i + (width * blockIdx.y * blockDim.y) + width * j;

	/* przypisanie odpowiednich wartosci pikseli do pamieci wspoldzielonej przez blok */
	block_in[i + 8 * j] = image_(type, index);

	float Gij = 0;
	/* obliczenie dyskretnej transoframy cosinusowej dla bloku*/
	for (int x = 0; x < 8; x++) {
		for (int y = 0; y < 8; y++) {
			converted = block_in[x + 8 * y];
			Gij += converted * cos_table[x][i] * cos_table[y][j];

		}
	}

	Gij = ((alpha(i)) * (alpha(j))) * Gij;

	/* kwantyzacja */
	float tmp;

	if (type == Y)
		tmp = Gij / fdtbl_Y[i + 8 * j];
	
	else
		tmp = Gij / fdtbl_Cb[i + 8 * j];


	/* przypisanie wartosci w kolejnosci zigzag */
	size_t block_nr = blockIdx.x + (gridDim.x * blockIdx.y);
	size_t pixel_nr = zigzag[i + 8 * j];
	result_(type, block_nr, pixel_nr) = (int16_t)(tmp);
}


int16_t *dct_CUDA(int8_t *Y_in, int8_t *Cb_in, int8_t *Cr_in, size_t width, size_t height, int *num_blocks, const float *dtY, const float *dtCb)
{
	int8_t *dev_image; /* zawiera 3 skladowe obrazka (Y, Cb, Cr)*/

	/* zmienne zawierja tablice 3 elemntowa dla 3 skladowych obrazka (Y, Cb, Cr)
		kazdej skladowej przypozadkowana jest tablica blokow
		a w kazdy blok jest tablica 64 elementowa	*/
	int16_t *dev_res;
	int16_t *dc_res;

	hipEvent_t start, stop; // pomiar czasu wykonania j�dra

	int16_t *host_tmp = (int16_t *)malloc(3 * width * height * sizeof(int16_t));
	size_t number_of_pixels = width * height;
	float elapsedTime = 0.0f;

	int grid_size_x = (int)(width / 8); /* liczba blokow watkow w sieci w kierunku x */
	int grid_size_y = (int)(height / 8); /* liczba blokow watkow w sieci w kierunku y */

	/* rozmiar siatki grid_size_x * grid_size_y * 3 skladowe obrazka (Y, Cb, Cr) */
	dim3 dimGrid(grid_size_x, grid_size_y, 3);
	/* rozmiar bloku watkow, zawieral bedzie 1 blok do obliczenia DCT*/
	dim3 dimBlock(8, 8);
	/* rozmiar pamieci wspoldzielonej przez 1 blok watkow */
	size_t shShize = (64 * sizeof(int8_t));

	checkCudaErrors(hipSetDevice(0));

	/* Alokacja pamieci */

	/* alokacja danych wejsciowych do urzadzenia */
	checkCudaErrors(hipMalloc((void **)&dev_image, 3 * (width * height * sizeof(int8_t))));

	/* alokacja dla danych wyjsciowych z urzadzenia*/
	checkCudaErrors(hipMalloc((void **)&dev_res, 3 * (width * height * sizeof(int16_t))));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(fdtbl_Y), (void *)dtY, (N) * sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(fdtbl_Cb), (void *)dtCb, (N) * sizeof(float)));

	/* kopiowanie pami�ci do urz�dzenia */
	checkCudaErrors(hipMemcpy(dev_image + (width * height * Y), 
		Y_in, width * height * sizeof(int8_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_image + (width * height * Cb), 
		Cb_in, width * height * sizeof(int8_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_image + (width * height * Cr), 
		Cr_in, width * height * sizeof(int8_t), hipMemcpyHostToDevice));

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));


	/* wywolanie funkcji jadra */
	dtf_kernel << < dimGrid, dimBlock, shShize >> > (dev_res, dev_image, 
														width, height, number_of_pixels);

	checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipDeviceSynchronize());


	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

	printf("GPU (kernel) time = %.3f ms \n",
		elapsedTime);


	/* Kopiowanie wynikow z pamieci urzadzenia do hosta */
	checkCudaErrors(hipMemcpy(host_tmp, dev_res , 3 * width * height * sizeof(int16_t), hipMemcpyDeviceToHost));


	/* zwolnienie pamieci */
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));
	hipFree(dev_image);
	hipFree(dev_res);

	checkCudaErrors(hipDeviceReset());

	return host_tmp;
}