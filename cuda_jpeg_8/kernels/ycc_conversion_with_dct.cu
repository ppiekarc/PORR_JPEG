#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include "ycc_conversion_with_dct.h"
#include <hip/hip_runtime.h>
#include <>
#include "hip/hip_runtime_api.h"
#include "helper_timer.h"

#define N 64
#define R_CHANNEL 0
#define G_CHANNEL 1
#define B_CHANNEL 2

#define Y_CHANNEL 0
#define Cb_CHANNEL 1
#define Cr_CHANNEL 2

__constant__ static uint8_t zigzag[64] = {
        0, 1, 5, 6,14,15,27,28,
        2, 4, 7,13,16,26,29,42,
        3, 8,12,17,25,30,41,43,
        9,11,18,24,31,40,44,53,
        10,19,23,32,39,45,52,54,
        20,22,33,38,46,51,55,60,
        21,34,37,47,50,56,59,61,
        35,36,48,49,57,58,62,63
};

__constant__ static float fdtbl_Y[N];
__constant__ static float fdtbl_Cb[N];

__constant__ static int32_t YR[256];
__constant__ static int32_t YG[256];
__constant__ static int32_t YB[256];

__constant__ static int32_t CbR[256];
__constant__ static int32_t CbG[256];
__constant__ static int32_t CbB[256];

__constant__ static int32_t CrR[256];
__constant__ static int32_t CrG[256];
__constant__ static int32_t CrB[256];

#define  Y(R,G,B) ((uint8_t)((YR[(R)] + YG[(G)] + YB[(B)]) >> 16 ) - 128)
#define Cb(R,G,B) ((uint8_t)((CbR[(R)] + CbG[(G)] + CbB[(B)]) >> 16 ))
#define Cr(R,G,B) ((uint8_t)((CrR[(R)] + CrG[(G)] + CrB[(B)]) >> 16 ))

#define image_(t, index) image[(t * width * height) + index]
#define result_(t, b, p) result[(t * width * height) + (b * 64) + p]

__global__ static void dtf_kernel(int16_t *result, uint8_t *image, size_t width, size_t height) {

    float tmp0, tmp1, tmp2, tmp3, tmp4, tmp5, tmp6, tmp7;
    float tmp10, tmp11, tmp12, tmp13;
    float z1, z2, z3, z4, z5, z11, z13;
    float *dataptr;
    __shared__ float datafloat[64];
    float temp;
    float *fdtbl;
    int8_t ctr;
    size_t j = threadIdx.x;
    size_t i = threadIdx.y;

    const unsigned int channel_type = blockIdx.z;
    if (i == 0) {
        for (size_t i = 0; i < 8; i++) {
            size_t index = (blockIdx.x * 8) + i + (width * blockIdx.y * 8) + width * j;
            if (channel_type == Y_CHANNEL) {
                datafloat[i + (8 * j)] = (int8_t)Y(image_(R_CHANNEL, index), image_(G_CHANNEL, index), image_(B_CHANNEL, index));
            } else if (channel_type == Cb_CHANNEL) {
                datafloat[i + (8 * j)] = (int8_t)Cb(image_(R_CHANNEL, index), image_(G_CHANNEL, index), image_(B_CHANNEL, index));
            } else {
                datafloat[i + (8 * j)] = (int8_t)Cr(image_(R_CHANNEL, index), image_(G_CHANNEL, index), image_(B_CHANNEL, index));
            }
        }

        /* Pass 1: process rows. */
        dataptr = datafloat + (j * 8);
        tmp0 = dataptr[0] + dataptr[7];
        tmp7 = dataptr[0] - dataptr[7];
        tmp1 = dataptr[1] + dataptr[6];
        tmp6 = dataptr[1] - dataptr[6];
        tmp2 = dataptr[2] + dataptr[5];
        tmp5 = dataptr[2] - dataptr[5];
        tmp3 = dataptr[3] + dataptr[4];
        tmp4 = dataptr[3] - dataptr[4];

        tmp10 = tmp0 + tmp3;
        tmp13 = tmp0 - tmp3;
        tmp11 = tmp1 + tmp2;
        tmp12 = tmp1 - tmp2;

        dataptr[0] = tmp10 + tmp11;
        dataptr[4] = tmp10 - tmp11;

        z1 = (tmp12 + tmp13) * ((float) 0.707106781);
        dataptr[2] = tmp13 + z1;
        dataptr[6] = tmp13 - z1;

        tmp10 = tmp4 + tmp5;
        tmp11 = tmp5 + tmp6;
        tmp12 = tmp6 + tmp7;

        z5 = (tmp10 - tmp12) * ((float) 0.382683433);
        z2 = ((float) 0.541196100) * tmp10 + z5;
        z4 = ((float) 1.306562965) * tmp12 + z5;
        z3 = tmp11 * ((float) 0.707106781);

        z11 = tmp7 + z3;
        z13 = tmp7 - z3;

        dataptr[5] = z13 + z2;
        dataptr[3] = z13 - z2;
        dataptr[1] = z11 + z4;
        dataptr[7] = z11 - z4;
    }
    /* Pass 2: process columns */
    if (i == 1) {
        dataptr = datafloat + j;

        tmp0 = dataptr[0] + dataptr[56];
        tmp7 = dataptr[0] - dataptr[56];
        tmp1 = dataptr[8] + dataptr[48];
        tmp6 = dataptr[8] - dataptr[48];
        tmp2 = dataptr[16] + dataptr[40];
        tmp5 = dataptr[16] - dataptr[40];
        tmp3 = dataptr[24] + dataptr[32];
        tmp4 = dataptr[24] - dataptr[32];

        tmp10 = tmp0 + tmp3;
        tmp13 = tmp0 - tmp3;
        tmp11 = tmp1 + tmp2;
        tmp12 = tmp1 - tmp2;

        dataptr[0] = tmp10 + tmp11;
        dataptr[32] = tmp10 - tmp11;

        z1 = (tmp12 + tmp13) * ((float) 0.707106781);
        dataptr[16] = tmp13 + z1;
        dataptr[48] = tmp13 - z1;

        tmp10 = tmp4 + tmp5;
        tmp11 = tmp5 + tmp6;
        tmp12 = tmp6 + tmp7;

        z5 = (tmp10 - tmp12) * ((float) 0.382683433);
        z2 = ((float) 0.541196100) * tmp10 + z5;
        z4 = ((float) 1.306562965) * tmp12 + z5;
        z3 = tmp11 * ((float) 0.707106781);

        z11 = tmp7 + z3;
        z13 = tmp7 - z3;
        dataptr[40] = z13 + z2;
        dataptr[24] = z13 - z2;
        dataptr[8] = z11 + z4;
        dataptr[56] = z11 - z4;

        fdtbl = (channel_type == 0) ? fdtbl_Y : fdtbl_Cb;

        for (size_t i = 0; i < 8; i++) {
            /* quantization and scaling factor */
            temp = datafloat[i + (8 * j)] * fdtbl[i + (8 * j)];
            /* Round to nearest integer. */
            size_t block_nr = blockIdx.x + (gridDim.x * blockIdx.y);
            size_t pixel_nr = zigzag[i + 8 * j];
            result_(channel_type, block_nr, pixel_nr) = (int16_t)((int16_t)(temp + 16384.5) - 16384);
        }
    }
}

int16_t *ycc_conversion_with_dct(uint8_t *R, uint8_t *G, uint8_t *B, size_t width, size_t height, int *num_blocks,
                                 const float *dtY, const float *dtCb, int32_t *tYR, int32_t *tYG,
                                 int32_t *tYB, int32_t *tCbR, int32_t *tCbG, int32_t *tCbB, int32_t *tCrR,
                                 int32_t *tCrG, int32_t *tCrB) {

    uint8_t *dev_image; /* zawiera 3 skladowe obrazka (Y, Cb, Cr)*/

    /* zmienne zawierja tablice 3 elemntowa dla 3 skladowych obrazka (Y, Cb, Cr)
    kazdej skladowej przypozadkowana jest tablica blokow
    a w kazdy blok jest tablica 64 elementowa	*/
    int16_t *dev_res;
    int16_t *host_result;

    hipEvent_t start, stop; // pomiar czasu wykonania j�dra
    float elapsedTime = 0.0f;

    int grid_size_x = (int)(width / 8); /* liczba blokow watkow w sieci w kierunku x */
    int grid_size_y = (int)(height / 8); /* liczba blokow watkow w sieci w kierunku y */

    /* rozmiar siatki grid_size_x * grid_size_y * 3 skladowe obrazka (Y, Cb, Cr) */
    dim3 dimGrid(grid_size_x, grid_size_y, 3);
    /* rozmiar bloku watkow, zawieral bedzie 1 blok do obliczenia DCT*/
    dim3 dimBlock(8, 2);
    /* rozmiar pamieci wspoldzielonej przez 1 blok watkow */
    size_t shShize = (64 * sizeof(int8_t));

    checkCudaErrors(hipSetDevice(0));

    /* alokacja danych wejsciowych do urzadzenia */
    checkCudaErrors(hipMalloc((void **)&dev_image, 3 * (width * height * sizeof(int8_t))));

    /* alokacja dla danych wyjsciowych z urzadzenia*/
    checkCudaErrors(hipMalloc((void **)&dev_res, 3 * (width * height * sizeof(int16_t))));

    /* alokacj dla danych wyjsciowych do hosta */
    host_result = (int16_t *)malloc(3 * width * height * sizeof(int16_t));

    /* przekopiowanie stalych tablic kwantyzacji do urzadzenia */
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(fdtbl_Y), (void *)dtY, (N) * sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(fdtbl_Cb), (void *)dtCb, (N) * sizeof(float)));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(YR), (void *)tYR, (256) * sizeof(int32_t)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(YB), (void *)tYB, (256) * sizeof(int32_t)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(YG), (void *)tYG, (256) * sizeof(int32_t)));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CbR), (void *)tCbR, (256) * sizeof(int32_t)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CbG), (void *)tCbG, (256) * sizeof(int32_t)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CbB), (void *)tCbB, (256) * sizeof(int32_t)));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CrR), (void *)tCrR, (256) * sizeof(int32_t)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CrG), (void *)tCrG, (256) * sizeof(int32_t)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CrB), (void *)tCrB, (256) * sizeof(int32_t)));

    /* kopiowanie pami�ci do urz�dzenia */
    checkCudaErrors(hipMemcpy(dev_image + (width * height * R_CHANNEL), R, width * height * sizeof(int8_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_image + (width * height * G_CHANNEL), G, width * height * sizeof(int8_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_image + (width * height * B_CHANNEL), B, width * height * sizeof(int8_t), hipMemcpyHostToDevice));

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));

    dtf_kernel << < dimGrid, dimBlock, shShize >> > (dev_res, dev_image, width, height);

    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipDeviceSynchronize());


    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

//	printf("GPU (kernel) time = %.3f ms \n",
//		elapsedTime);

    /* Kopiowanie wynikow z pamieci urzadzenia do hosta */
    checkCudaErrors(hipMemcpy(host_result, dev_res, 3 * width * height * sizeof(int16_t), hipMemcpyDeviceToHost));

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    hipFree(dev_image);
    hipFree(dev_res);

    checkCudaErrors(hipDeviceReset());

    return host_result;
}